
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA Kernel function to add the elements of two arrays
__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    const int N = 1000; // Array size
    int size = N * sizeof(int);

    // Host arrays
    int h_a[N], h_b[N], h_c[N];

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Device arrays
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with N threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print some of the results
    for (int i = 0; i < 10; i++) {
        std::cout << "h_a[" << i << "] + h_b[" << i << "] = " << h_c[i] << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
